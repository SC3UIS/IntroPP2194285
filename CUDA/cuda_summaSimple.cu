// From https://www.programiz.com/c-programming/c-for-loop 
// Modified by Fabian Sanchez and Wilmer Farfan for learning purposes 2023
// Simple Program to calculate the sum of first n natural numbers parallelized with CUDA
// Positive integers 1,2,3...n are known as natural numbers

#include <stdio.h>
#include <time.h>

// Incluir la biblioteca de CUDA
#include <hip/hip_runtime.h>
// Definir la función de kernel
__global__ void sumNumbers(int *sum, int num)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx <= num)
    {
        atomicAdd(sum, idx);
    }
}
int main()
{
    int num, sum = 0;
    int *d_sum; // Puntero para la suma en el dispositivo

    // Contabilizar el tiempo    
    clock_t startTime, endTime;
  
    // Se empieza a contabilizar el tiempo
    startTime = clock();
    
    printf("\nSuma de los primeros n numeros naturales (Paralelizado con CUDA)\n");
    printf("Ingrese un Entero Positivo: ");
    scanf("%d", &num);

    // Reservar memoria en el dispositivo para la suma
    hipMalloc((void **)&d_sum, sizeof(int));
    // Copiar la suma inicial al dispositivo
    hipMemcpy(d_sum, &sum, sizeof(int), hipMemcpyHostToDevice);

    // Definir el tamaño de los bloques y la cantidad de bloques
    int blockSize = 256;
    int gridSize = (num + blockSize - 1) / blockSize;

    // Lanzar el kernel en la GPU
    sumNumbers<<<gridSize, blockSize>>>(d_sum, num);

    // Copiar la suma resultante desde el dispositivo al host
    hipMemcpy(&sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);

    // Liberar memoria en el dispositivo
    hipFree(d_sum);

    // Se termina de contabilizar el tiempo
    endTime = clock(); 

    printf("Suma Total = %d\n", sum);    
    printf("La ejecucion tardo %lf segundos\n", (double)(endTime - startTime) / CLOCKS_PER_SEC);

    return 0;
}
